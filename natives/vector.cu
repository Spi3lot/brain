#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <jni.h>

#define CHECK_CUDA_ERROR(call)                                                           \
    {                                                                                    \
        const hipError_t error = call;                                                  \
        if (error != hipSuccess)                                                        \
        {                                                                                \
            fprintf(stderr, "CUDA Error: %s:%d, ", __FILE__, __LINE__);                  \
            fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error)); \
            return;                                                                      \
        }                                                                                \
    }

#define BLOCK_SIZE 256

/**
 *
 *  Kernel functions for vector operations
 *
 */

// Negate Kernel
__global__ void negateKernel(float *d_input, float *d_output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        d_output[idx] = -d_input[idx];
    }
}

// Add Kernel
__global__ void addKernel(float *d_A, float *d_B, float *d_C, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        d_C[idx] = d_A[idx] + d_B[idx];
    }
}

// Sub Kernel
__global__ void subKernel(float *d_A, float *d_B, float *d_C, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        d_C[idx] = d_A[idx] - d_B[idx];
    }
}

// Mult Scalar Kernel
__global__ void multScalarKernel(float *d_input, float factor, float *d_output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        d_output[idx] = d_input[idx] * factor;
    }
}

// Mult Vector Kernel
__global__ void multVectorKernel(float *d_A, float *d_B, float *d_C, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        d_C[idx] = d_A[idx] * d_B[idx];
    }
}

// Matrix Multiplication Kernel
__global__ void matMulKernel(float *d_A, float *d_B, float *d_C, int A_rows, int A_cols, int B_cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < A_rows && col < B_cols)
    {
        float value = 0;
        for (int k = 0; k < A_cols; ++k)
        {
            value += d_A[row * A_cols + k] * d_B[k * B_cols + col];
        }
        d_C[row * B_cols + col] = value;
    }
}

// Dot Product Kernel with Reduction
__global__ void dotKernel(float *d_A, float *d_B, float *d_result, int size)
{
    __shared__ float temp[BLOCK_SIZE];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int threadId = threadIdx.x;

    float sum = 0;
    if (idx < size)
    {
        sum = d_A[idx] * d_B[idx];
    }

    temp[threadId] = sum;
    __syncthreads();

    // Reduction to sum all the elements
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) // this only
    {
        if (threadId < stride)
        {
            temp[threadId] += temp[threadId + stride];
        }
        __syncthreads();
    }

    if (threadId == 0)
    {
        atomicAdd(d_result, temp[0]);
    }
}

/**
 *
 *  Helper functions for common tasks
 *
 */

// Error checking helper
void checkCudaError(hipError_t error, const char *msg)
{
    if (error != hipSuccess)
    {
        printf("CUDA Error: %s - %s\n", msg, hipGetErrorString(error));
        hipDeviceReset();
    }
}

// Allocate memory and copy data from host to device
float *allocateJfloatArrayAndCopyToDevice(JNIEnv *env, jfloatArray jData, int size)
{
    jfloat *h_data = env->GetFloatArrayElements(jData, 0);
    float *d_data;
    checkCudaError(hipMalloc(&d_data, size * sizeof(float)), "Failed to allocate GPU memory");
    checkCudaError(hipMemcpy(d_data, h_data, size * sizeof(float), hipMemcpyHostToDevice), "Failed to copy data to GPU");
    env->ReleaseFloatArrayElements(jData, h_data, 0);
    return d_data;
}

// Allocate memory and copy data from host to device
float *allocateJobjectArrayAndCopyToDevice(JNIEnv *env, jobjectArray jData, int rows, int cols)
{
    int size = rows * cols;
    float *h_data = new float[size];

    for (int i = 0; i < rows; ++i)
    {
        jfloatArray rowArray = (jfloatArray)env->GetObjectArrayElement(jData, i);
        jfloat *rowData = env->GetFloatArrayElements(rowArray, 0);
        for (int j = 0; j < cols; ++j)
        {
            h_data[i * cols + j] = rowData[j];
        }
        env->ReleaseFloatArrayElements(rowArray, rowData, 0);
        env->DeleteLocalRef(rowArray);
    }

    float *d_data;
    checkCudaError(hipMalloc(&d_data, size * sizeof(float)), "Failed to allocate GPU memory");
    checkCudaError(hipMemcpy(d_data, h_data, size * sizeof(float), hipMemcpyHostToDevice), "Failed to copy data to GPU");

    delete[] h_data;
    return d_data;
}

// Copy data from device to host and return it as a new float array
jfloatArray copyDeviceToHostAndCreateArray(JNIEnv *env, float *d_data, int size)
{
    jfloatArray resultArray = env->NewFloatArray(size);
    jfloat *h_output = new jfloat[size];
    checkCudaError(hipMemcpy(h_output, d_data, size * sizeof(float), hipMemcpyDeviceToHost), "Failed to copy result to host");
    env->SetFloatArrayRegion(resultArray, 0, size, h_output);
    delete[] h_output;
    return resultArray;
}

// Utility to allocate device memory and check for errors
float *allocateDeviceMemory(JNIEnv *env, jfloatArray array, int size)
{
    jfloat *hostArray = env->GetFloatArrayElements(array, NULL);
    if (hostArray == NULL)
    {
        fprintf(stderr, "Error: Failed to get array elements from Java.\n");
        return nullptr;
    }

    float *deviceArray;
    const hipError_t mallocError = hipMalloc((void **)&deviceArray, size * sizeof(float));
    if (mallocError != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s:%d, code: %d, reason: %s\n", __FILE__, __LINE__, mallocError, hipGetErrorString(mallocError));
        return nullptr;
    }

    const hipError_t memcpyError = hipMemcpy(deviceArray, hostArray, size * sizeof(float), hipMemcpyHostToDevice);
    if (memcpyError != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s:%d, code: %d, reason: %s\n", __FILE__, __LINE__, memcpyError, hipGetErrorString(memcpyError));
        hipFree(deviceArray); // Clean up memory in case of failure
        return nullptr;
    }

    env->ReleaseFloatArrayElements(array, hostArray, 0);
    return deviceArray; // Return the allocated and populated device array
}

// Utility to copy memory back to host and release device memory
void releaseDeviceMemory(JNIEnv *env, jfloatArray result, float *deviceArray, int size)
{
    jfloat *hostResult = env->GetFloatArrayElements(result, NULL);
    CHECK_CUDA_ERROR(hipMemcpy(hostResult, deviceArray, size * sizeof(float), hipMemcpyDeviceToHost));
    env->ReleaseFloatArrayElements(result, hostResult, 0);
    hipFree(deviceArray);
}

/**
 *
 *  JNI functions for vector operations
 *
 */
extern "C"
{
    // Negate
    JNIEXPORT jobject JNICALL Java_brain_math_GpuVector_negate(JNIEnv *env, jobject obj)
    {
        // Retrieve the size field and values array from the vector
        jclass vectorClass = env->GetObjectClass(obj);
        jfieldID valuesField = env->GetFieldID(vectorClass, "values", "[F");
        jfloatArray jData = (jfloatArray)env->GetObjectField(obj, valuesField);
        jint size = env->GetArrayLength(jData);

        // Allocate GPU memory and copy data from the vector
        float *d_A = allocateJfloatArrayAndCopyToDevice(env, jData, size);
        float *d_C;
        checkCudaError(hipMalloc(&d_C, size * sizeof(float)), "Failed to allocate output memory");

        // Launch kernel
        int gridSize = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        negateKernel<<<gridSize, BLOCK_SIZE>>>(d_A, d_C, size);

        // Copy result back to host
        jfloatArray resultArray = copyDeviceToHostAndCreateArray(env, d_C, size);

        // Free device memory
        hipFree(d_A);
        hipFree(d_C);
        hipDeviceReset();

        // Create and return new GpuVector with the result
        jobject resultVector = env->NewObject(vectorClass, env->GetMethodID(vectorClass, "<init>", "(I)V"), size);
        env->SetObjectField(resultVector, valuesField, resultArray);
        return resultVector;
    }

    // Add
    JNIEXPORT jobject JNICALL Java_brain_math_GpuVector_add(JNIEnv *env, jobject obj, jobject v)
    {
        // Retrieve the size field and values array from both vectors
        jclass vectorClass = env->GetObjectClass(obj);
        jfieldID sizeField = env->GetFieldID(vectorClass, "values", "[F");
        jint size = env->GetArrayLength((jfloatArray)env->GetObjectField(obj, sizeField));

        jfloatArray jData1 = (jfloatArray)env->GetObjectField(obj, sizeField);
        jfloatArray jData2 = (jfloatArray)env->GetObjectField(v, sizeField);

        // Allocate GPU memory and copy data from both vectors
        float *d_A = allocateJfloatArrayAndCopyToDevice(env, jData1, size);
        float *d_B = allocateJfloatArrayAndCopyToDevice(env, jData2, size);
        float *d_C;
        checkCudaError(hipMalloc(&d_C, size * sizeof(float)), "Failed to allocate output memory");

        // Launch kernel
        int gridSize = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        addKernel<<<gridSize, BLOCK_SIZE>>>(d_A, d_B, d_C, size);

        // Copy result back to host
        jfloatArray resultArray = copyDeviceToHostAndCreateArray(env, d_C, size);

        // Free device memory
        hipDeviceReset();

        // Create and return new GpuVector with the result
        jobject resultVector = env->NewObject(vectorClass, env->GetMethodID(vectorClass, "<init>", "(I)V"), size);
        env->SetObjectField(resultVector, sizeField, resultArray);
        return resultVector;
    }

    // Sub
    JNIEXPORT jobject JNICALL Java_brain_math_GpuVector_sub(JNIEnv *env, jobject obj, jobject v)
    {
        // Retrieve the size field and values array from both vectors
        jclass vectorClass = env->GetObjectClass(obj);
        jfieldID sizeField = env->GetFieldID(vectorClass, "values", "[F");
        jint size = env->GetArrayLength((jfloatArray)env->GetObjectField(obj, sizeField));

        jfloatArray jData1 = (jfloatArray)env->GetObjectField(obj, sizeField);
        jfloatArray jData2 = (jfloatArray)env->GetObjectField(v, sizeField);

        // Allocate GPU memory and copy data from both vectors
        float *d_A = allocateJfloatArrayAndCopyToDevice(env, jData1, size);
        float *d_B = allocateJfloatArrayAndCopyToDevice(env, jData2, size);
        float *d_C;
        checkCudaError(hipMalloc(&d_C, size * sizeof(float)), "Failed to allocate output memory");

        // Launch kernel
        int gridSize = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        subKernel<<<gridSize, BLOCK_SIZE>>>(d_A, d_B, d_C, size);

        // Copy result back to host
        jfloatArray resultArray = copyDeviceToHostAndCreateArray(env, d_C, size);

        // Free device memory
        hipDeviceReset();

        // Create and return new GpuVector with the result
        jobject resultVector = env->NewObject(vectorClass, env->GetMethodID(vectorClass, "<init>", "(I)V"), size);
        env->SetObjectField(resultVector, sizeField, resultArray);
        return resultVector;
    }

    // Multiply by Scalar
    JNIEXPORT jobject JNICALL Java_brain_math_GpuVector_mult__F(JNIEnv *env, jobject obj, jfloat factor)
    {
        // Retrieve the size field and values array from the vector
        jclass vectorClass = env->GetObjectClass(obj);
        jfieldID sizeField = env->GetFieldID(vectorClass, "values", "[F");
        jint size = env->GetArrayLength((jfloatArray)env->GetObjectField(obj, sizeField));

        jfloatArray jData = (jfloatArray)env->GetObjectField(obj, sizeField);

        // Allocate GPU memory and copy data from the vector
        float *d_A = allocateJfloatArrayAndCopyToDevice(env, jData, size);
        float *d_C;
        checkCudaError(hipMalloc(&d_C, size * sizeof(float)), "Failed to allocate output memory");

        // Launch kernel
        int gridSize = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        multScalarKernel<<<gridSize, BLOCK_SIZE>>>(d_A, factor, d_C, size);

        // Copy result back to host
        jfloatArray resultArray = copyDeviceToHostAndCreateArray(env, d_C, size);

        // Free device memory
        hipDeviceReset();

        // Create and return new GpuVector with the result
        jobject resultVector = env->NewObject(vectorClass, env->GetMethodID(vectorClass, "<init>", "(I)V"), size);
        env->SetObjectField(resultVector, sizeField, resultArray);
        return resultVector;
    }

    // Multiply by Vector
    JNIEXPORT jobject JNICALL Java_brain_math_GpuVector_mult__Lbrain_math_Vector_2(JNIEnv *env, jobject obj, jobject v)
    {
        // Retrieve the size field and values array from both vectors
        jclass vectorClass = env->GetObjectClass(obj);
        jfieldID sizeField = env->GetFieldID(vectorClass, "values", "[F");
        jint size = env->GetArrayLength((jfloatArray)env->GetObjectField(obj, sizeField));

        jfloatArray jData1 = (jfloatArray)env->GetObjectField(obj, sizeField);
        jfloatArray jData2 = (jfloatArray)env->GetObjectField(v, sizeField);

        // Allocate GPU memory and copy data from both vectors
        float *d_A = allocateJfloatArrayAndCopyToDevice(env, jData1, size);
        float *d_B = allocateJfloatArrayAndCopyToDevice(env, jData2, size);
        float *d_C;
        checkCudaError(hipMalloc(&d_C, size * sizeof(float)), "Failed to allocate output memory");

        // Launch kernel
        int gridSize = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        multVectorKernel<<<gridSize, BLOCK_SIZE>>>(d_A, d_B, d_C, size);

        // Copy result back to host
        jfloatArray resultArray = copyDeviceToHostAndCreateArray(env, d_C, size);

        // Free device memory
        hipDeviceReset();

        // Create and return new GpuVector with the result
        jobject resultVector = env->NewObject(vectorClass, env->GetMethodID(vectorClass, "<init>", "(I)V"), size);
        env->SetObjectField(resultVector, sizeField, resultArray);
        return resultVector;
    }

    JNIEXPORT jobject JNICALL Java_brain_math_GpuVector_mult__Lbrain_math_Matrix_2(JNIEnv *env, jobject obj, jobject m)
    {
        // Retrieve the size field and values array from both matrices
        jclass matrixClass = env->GetObjectClass(obj);
        jfieldID valuesField = env->GetFieldID(matrixClass, "values", "[Lbrain/math/Vector;");
        jobjectArray jData1 = (jobjectArray)env->GetObjectField(obj, valuesField);
        jobjectArray jData2 = (jobjectArray)env->GetObjectField(m, valuesField);

        jint A_rows = env->GetArrayLength(jData1);
        jint A_cols = env->GetArrayLength((jfloatArray)env->GetObjectArrayElement(jData1, 0));
        jint B_rows = env->GetArrayLength(jData2);
        jint B_cols = env->GetArrayLength((jfloatArray)env->GetObjectArrayElement(jData2, 0));

        if (A_cols != B_rows)
        {
            jclass illegalArgumentException = env->FindClass("java/lang/IllegalArgumentException");
            env->ThrowNew(illegalArgumentException, "Matrix column amount must match");
            return nullptr;
        }

        // Allocate GPU memory and copy data from both matrices
        float *d_A = allocateJobjectArrayAndCopyToDevice(env, jData1, A_rows, A_cols);
        float *d_B = allocateJobjectArrayAndCopyToDevice(env, jData2, B_rows, B_cols);
        float *d_C;
        checkCudaError(hipMalloc(&d_C, A_rows * B_cols * sizeof(float)), "Failed to allocate GPU memory for result");

        // Launch kernel
        dim3 blockSize(16, 16);
        dim3 gridSize((B_cols + blockSize.x - 1) / blockSize.x, (A_rows + blockSize.y - 1) / blockSize.y);
        matMulKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, A_rows, A_cols, B_cols);

        // Copy result back to host
        float *h_C = new float[A_rows * B_cols];
        hipMemcpy(h_C, d_C, A_rows * B_cols * sizeof(float), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        // Reset the device
        hipDeviceReset();

        // Create a new GpuVector object for each row in the resulting matrix
        jclass vectorClass = env->FindClass("brain/math/GpuVector");
        jmethodID vectorConstructor = env->GetMethodID(vectorClass, "<init>", "(I)V");
        jobjectArray resultArray = env->NewObjectArray(A_rows, vectorClass, nullptr);

        for (int i = 0; i < A_rows; ++i)
        {
            jfloatArray rowArray = env->NewFloatArray(B_cols);
            env->SetFloatArrayRegion(rowArray, 0, B_cols, h_C + i * B_cols);
            jobject rowVector = env->NewObject(vectorClass, vectorConstructor, B_cols);
            env->SetObjectField(rowVector, valuesField, rowArray);
            env->SetObjectArrayElement(resultArray, i, rowVector);
        }

        delete[] h_C;

        // Create and return new GpuMatrix with the result
        jobject resultMatrix = env->NewObject(matrixClass, env->GetMethodID(matrixClass, "<init>", "(II)V"), B_cols, A_rows);
        env->SetObjectField(resultMatrix, valuesField, resultArray);
        return resultMatrix;
    }

    // Division by Scalar
    JNIEXPORT jobject JNICALL Java_brain_math_GpuVector_div(JNIEnv *env, jobject obj, jfloat divisor)
    {
        return Java_brain_math_GpuVector_mult__F(env, obj, 1.0f / divisor); // Reuse scalar mult method with reciprocal divisor
    }

    // Dot Product
    JNIEXPORT jfloat JNICALL Java_brain_math_GpuVector_dot(JNIEnv *env, jobject obj, jobject v)
    {
        // Retrieve the size field and values array from both vectors
        jclass vectorClass = env->GetObjectClass(obj);
        jfieldID sizeField = env->GetFieldID(vectorClass, "values", "[F");
        jint size = env->GetArrayLength((jfloatArray)env->GetObjectField(obj, sizeField));

        jfloatArray jData1 = (jfloatArray)env->GetObjectField(obj, sizeField);
        jfloatArray jData2 = (jfloatArray)env->GetObjectField(v, sizeField);

        // Allocate GPU memory and copy data from both vectors
        float *d_A = allocateJfloatArrayAndCopyToDevice(env, jData1, size);
        float *d_B = allocateJfloatArrayAndCopyToDevice(env, jData2, size);
        float *d_result;
        checkCudaError(hipMalloc(&d_result, sizeof(float)), "Failed to allocate result memory");

        checkCudaError(hipMemset(d_result, 0, sizeof(float)), "Failed to zero out result");

        // Launch kernel
        int gridSize = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dotKernel<<<gridSize, BLOCK_SIZE>>>(d_A, d_B, d_result, size);

        // Copy result back to host
        float result;
        checkCudaError(hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost), "Failed to copy result to host");

        // Free device memory
        hipDeviceReset();

        return result;
    }
}